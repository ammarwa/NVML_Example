#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti_events.h>
#include "simple.c"

#include <unistd.h>
#include <pthread.h>

#define CHECK_CU_ERROR(err, cufunc)                                     \
if (err != hipSuccess)                                              \
 {                                                                   \
   printf ("Error %d for CUDA Driver API function '%s'.\n",          \
           err, cufunc);                                             \
   exit(-1);                                                         \
 }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                       \
if (err != CUPTI_SUCCESS)                                     \
 {                                                           \
   const char *errstr;                                       \
   cuptiGetResultString(err, &errstr);                       \
   printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",  \
           __FILE__, __LINE__, errstr, cuptifunc);           \
   exit(-1);                                                 \
 }

#define EVENT_NAME "inst_executed"

static volatile int testComplete = 0;

static hipCtx_t context;
static hipDevice_t device;
static const char *eventName;

void *
sampling_func(void *arg)
{
CUptiResult cuptiErr;
CUpti_EventGroup eventGroup;
CUpti_EventID eventId;
size_t bytesRead, valueSize;
uint32_t numInstances = 0, j = 0;
uint64_t *eventValues = NULL, eventVal = 0;
uint32_t profile_all = 1;

cuptiErr = cuptiSetEventCollectionMode(context,
                                      CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");

cuptiErr = cuptiEventGroupCreate(context, &eventGroup, 0);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

cuptiErr = cuptiEventGetIdFromName(device, eventName, &eventId);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGetIdFromName");

cuptiErr = cuptiEventGroupAddEvent(eventGroup, eventId);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

cuptiErr = cuptiEventGroupSetAttribute(eventGroup,
                                      CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                                      sizeof(profile_all), &profile_all);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupSetAttribute");

cuptiErr = cuptiEventGroupEnable(eventGroup);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");

valueSize = sizeof(numInstances);
cuptiErr = cuptiEventGroupGetAttribute(eventGroup,
                                      CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
                                      &valueSize, &numInstances);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupGetAttribute");

bytesRead = sizeof(uint64_t) * numInstances;
eventValues = (uint64_t *) malloc(bytesRead);
if (eventValues == NULL) {
   printf("%s:%d: Failed to allocate memory.\n", __FILE__, __LINE__);
   exit(-1);
}

while (!testComplete) {
 cuptiErr = cuptiEventGroupReadEvent(eventGroup,
                                     CUPTI_EVENT_READ_FLAG_NONE,
                                     eventId, &bytesRead, eventValues);
 CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");
 if (bytesRead != (sizeof(uint64_t) * numInstances)) {
   printf("Failed to read value for \"%s\"\n", eventName);
   exit(-1);
 }

 for (j = 0; j < numInstances; j++) {
   eventVal += eventValues[j];
 }
 printf("%s: %llu\n", eventName, (unsigned long long)eventVal);
//  usleep(10);
}

cuptiErr = cuptiEventGroupDisable(eventGroup);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");

cuptiErr = cuptiEventGroupDestroy(eventGroup);
CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

free(eventValues);
return NULL;
}

int
main(int argc, char *argv[])
{
int status;
pthread_t pThread;
hipError_t err;
int deviceNum;
int deviceCount;
char deviceName[32];

printf("Usage: %s [device_num] [event_name]\n", argv[0]);

err = hipInit(0);
CHECK_CU_ERROR(err, "hipInit");

err = hipGetDeviceCount(&deviceCount);
CHECK_CU_ERROR(err, "hipGetDeviceCount");

if (deviceCount == 0) {
 printf("There is no device supporting CUDA.\n");
 exit(-1);
}

if (argc > 1)
 deviceNum = atoi(argv[1]);
else
 deviceNum = 0;
printf("CUDA Device Number: %d\n", deviceNum);

err = hipDeviceGet(&device, deviceNum);
CHECK_CU_ERROR(err, "hipDeviceGet");

err = hipDeviceGetName(deviceName, 32, device);
CHECK_CU_ERROR(err, "hipDeviceGetName");

printf("CUDA Device Name: %s\n", deviceName);

if (argc > 2) {
 eventName = argv[2];
}
else {
 eventName = EVENT_NAME;
}

err = hipCtxCreate(&context, 0, device);
CHECK_CU_ERROR(err, "hipCtxCreate");


testComplete = 0;

// printf("Creating sampling thread\n");

status = pthread_create(&pThread, NULL, sampling_func, NULL);
if (status != 0) {
 perror("pthread_create");
 exit(-1);
}

func(5000);
// usleep(500000);
testComplete = 1;
pthread_join(pThread, NULL);

hipDeviceSynchronize();
return 0;
}
